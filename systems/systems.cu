#include "systems.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hiprtc.h>
#include <hip/hip_cooperative_groups.h>

namespace chameleon_1_cd {
    __device__ void calculateDiscreteModel(double* X, const double* a, const double h)
{
     
}

}

namespace rossler_cd {
    __device__ void calculateDiscreteModel(double* X, const double* a, const double h) {

    }
}
