#include "hip/hip_runtime.h"
#include <LLECUDA.cuh>
#include <string>
#include <iostream>
#include <fstream>
#include <hip/hiprtc.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_cooperative_groups.h>
namespace LLE_constants{
	
__device__ void calculateDiscreteModel(double *X, const double *a, const double h)
{
    double cos_term = cos(a[5] * X[1]); // Reuse cosine term

    // First phase updates
    X[0] += d_h1 * (-a[6] * X[1]);
    X[1] += d_h1 * (a[6] * X[0] + a[1] * X[2]);
    X[2] += d_h1 * (a[2] - a[3] * X[2] + a[4] * cos_term);

    // Second phase updates      // Compute d_h2 when needed
    X[2] = (X[2] + d_h2 * (a[2] + a[4] * cos_term)) / (1 + a[3] * d_h2);
    X[1] += d_h2 * (a[6] * X[0] + a[1] * X[2]);
    X[0] += d_h2 * (-a[6] * X[1]);
}

__device__ bool loopCalculateDiscreteModel(double *x, const double *params,
                                                    const int amountOfIterations)
{
    for (int i = 0; i < amountOfIterations; ++i)
    {

        calculateDiscreteModel(x, params, d_h);
    }
    return true;
}

namespace cg = cooperative_groups;
__global__ void calculateSystem(
    double* X,
    double* params,
    const double *paramLinspaceA,
    const double *paramLinspaceB,
    double **result
){
   extern __shared__ double sh_mem[];
    cg::thread_block block = cg::this_thread_block();
    cg::thread_group pair = cg::tiled_partition<2>(block); // Разделяем блок на группы по 2 потока

    const int idx_a = (threadIdx.x + blockIdx.x * blockDim.x) / 2;  
    const int idx_b = threadIdx.y + blockIdx.y * blockDim.y;
    if (idx_a >= d_size_linspace_A || idx_b >= d_size_linspace_B) return;

    const bool is_main_thread = pair.thread_rank() == 0; // Основной поток (0) или perturbated (1)

    const int thread_id = threadIdx.y * (blockDim.x / 2) + (threadIdx.x / 2);
    const int total_size_per_pair = 2 * d_XSize + d_paramsSize;  // Память для X, perturbated_X и params

    // Разделяем память между потоками в паре
    double* my_sh_X = &sh_mem[thread_id * total_size_per_pair];              
    double* my_sh_params = &sh_mem[thread_id * total_size_per_pair + d_XSize]; 
    double* my_sh_perturbated_X = &sh_mem[thread_id * total_size_per_pair + d_XSize + d_paramsSize]; 

    if (is_main_thread) {
        for (int i = 0; i < d_XSize; ++i) my_sh_X[i] = X[i];
        for (int i = 0; i < d_paramsSize; ++i) my_sh_params[i] = params[i];

        my_sh_params[d_idxParamA] = paramLinspaceA[idx_a];
        my_sh_params[d_idxParamB] = paramLinspaceB[idx_b];

        loopCalculateDiscreteModel(my_sh_X, my_sh_params, d_amountOfTransPoints);

        // Генератор случайных чисел
        hiprandState_t state;
        hiprand_init(idx_a, 0, 0, &state);

        double norm_factor = 0.0;
        for (int i = 0; i < d_XSize; ++i) {
            double z = hiprand_uniform(&state) - 0.5;
            norm_factor += z * z;
        }
        norm_factor = sqrt(norm_factor);

        for (int i = 0; i < d_XSize; ++i) {
            double z = (hiprand_uniform(&state) - 0.5) / norm_factor;
            my_sh_perturbated_X[i] = my_sh_X[i] + z * d_eps;
        }
    }

    pair.sync(); // Синхронизация внутри группы из 2 потоков

    double local_result = 0.0;  
    const double inv_eps = 1.0 / d_eps;

    for (int i = 0; i <= d_amountOfCalcBlocks; ++i) {
        if (is_main_thread) {
            loopCalculateDiscreteModel(my_sh_X, my_sh_params, d_amountOfNTPoints);
        } else {
            loopCalculateDiscreteModel(my_sh_perturbated_X, my_sh_params, d_amountOfNTPoints);
        }

        pair.sync(); // Синхронизация только внутри пары

        if (is_main_thread) {
            double distance = 0.0;
            for (int l = 0; l < d_XSize; ++l) {
                double diff = (my_sh_X[l] - my_sh_perturbated_X[l]) * inv_eps;
                distance += diff * diff;
            }
            distance = sqrt(distance);
            local_result += log(distance);

            for (int j = 0; j < d_XSize; ++j) {
                my_sh_perturbated_X[j] = my_sh_X[j] - ((my_sh_X[j] - my_sh_perturbated_X[j]) / distance);
            }
        }

        pair.sync(); // Синхронизация перед новой итерацией
    }

    if (is_main_thread) {
        atomicAdd(&result[idx_a][idx_b], local_result);
    }
}

__host__ double* linspace(double start, double end, int num)
{
    // Allocate memory for num doubles
    double* result = new double[num];

    // Handle edge cases
    if (num < 0)
    {
        delete[] result;  // Clean up before throwing
        throw std::invalid_argument("received negative number of points");
    }
    if (num == 0)
    {
        return result;  // Return empty array
    }
    if (num == 1)
    {
        result[0] = start;  // Assign single value
        return result;
    }

    // Calculate step size
    double step = (end - start) / (num - 1);

    // Fill the array
    for (int i = 0; i < num; ++i)
    {
        result[i] = start + i * step;
    }

    return result;
}

__host__ void LLE2D(
	const double tMax,
	const double NT,
	const double h,
	const double eps,
	const double transientTime,
	const double* initialConditions,
	const int amount_init,
	const double* params,
	const int amount_params,
	const double* linspaceA_params,
	const double* linspaceB_params,
	const int* indicesOfMutVars,
	std::string		OUT_FILE_PATH)
{

	double* linspaceA = linspace(linspaceA_params[0], linspaceA_params[1], linspaceA_params[2]);
	double* linspaceB = linspace(linspaceB_params[0], linspaceB_params[1], linspaceB_params[2]);

	int amountOfNTPoints = static_cast<int>(NT / h);
	int amountOfTransPoints= static_cast<int>(transientTime / h);
	int amountOfAllPoints= static_cast<int>(tMax / h);

	size_t freeMemory;
	size_t totalMemory;

	gpuErrorCheck(hipMemGetInfo(&freeMemory, &totalMemory));

	const int size_A =  static_cast<int>(linspaceA_params[2]);
	const int size_B =  static_cast<int>(linspaceB_params[2]);
	int NT_steps = static_cast<int>(NT/h);

	int amount_of_calc_blocks = static_cast<int>(amountOfAllPoints/amountOfNTPoints) + 1;

	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_idxParamA), &indicesOfMutVars[0], sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_idxParamB), &indicesOfMutVars[1], sizeof(int)));

	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_size_linspace_A), &size_A, sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_size_linspace_B), &size_B, sizeof(int)));

	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_h), &h, sizeof(double)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_transTime), &transientTime, sizeof(double)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_Nt_steps), &NT_steps, sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_paramsSize), &amount_params, sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_XSize), &amount_init, sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_amountOfNTPoints), &amountOfNTPoints, sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_amountOfTransPoints), &amountOfTransPoints, sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_amountOfAllpoints), &amountOfAllPoints, sizeof(int)));
    gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_amountOfCalcBlocks), &amount_of_calc_blocks, sizeof(int)));

    double h_h1 = params[0] * h;
    double h_h2 = (1 - params[0]) * h;
    gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_h1), &h_h1, sizeof(double)));
    gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_h2), &h_h2, sizeof(double)));
    gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_eps), &eps, sizeof(double)));


    int max_threads_y = 10; 
    int max_threads_x = 2 * max_threads_y;  

    int gridSizeY = (size_B + max_threads_y - 1) / max_threads_y;
    int gridSizeX = 2 * gridSizeY;

    // Ensure gridSizeX covers size_A with threadsPerBlock.x
    int minGridSizeX = (size_A + max_threads_x - 1) / max_threads_x;
    if (gridSizeX < minGridSizeX) {
        gridSizeX = minGridSizeX;  // At least enough blocks to cover size_A
    }

    // Define thread block and grid dimensions
    dim3 threadsPerBlock(max_threads_x, max_threads_y, 1);  // e.g., (16, 8, 1)
    dim3 blocksPerGrid(gridSizeX, gridSizeY, 1);
    size_t sharedMemSize = (max_threads_x * max_threads_y / 2) * (amount_params + amount_init + amount_init) * sizeof(double);  // For thread pairs
    printf("Total shared memory: %zu bytes\n", sharedMemSize);

    double** d_result;
    double** h_result_temp = new double*[size_A];
    gpuErrorCheck(hipMalloc(&d_result, size_A * sizeof(double*)));
    for (int i = 0; i < size_A; ++i) {
        gpuErrorCheck(hipMalloc(&h_result_temp[i], size_B * sizeof(double)));
        double zero = 0.0;

        for (int j = 0; j < size_B; ++j) {
            gpuErrorCheck(hipMemcpy(h_result_temp[i] + j, &zero, sizeof(double), hipMemcpyHostToDevice));
        }
    }
    gpuErrorCheck(hipMemcpy(d_result, h_result_temp, size_A * sizeof(double*), hipMemcpyHostToDevice));


	double* d_paramLinspaceA;
	double* d_paramLinspaceB;
	double* d_X;
	double* d_params;

	gpuErrorCheck(hipMalloc(&d_X, amount_init * sizeof(double)));
	gpuErrorCheck(hipMalloc(&d_params, amount_params * sizeof(double)));
	gpuErrorCheck(hipMalloc(&d_paramLinspaceA, size_A * sizeof(double)));
	gpuErrorCheck(hipMalloc(&d_paramLinspaceB, size_B * sizeof(double)));

	gpuErrorCheck(hipMemcpy(d_X, initialConditions, amount_init * sizeof(double),
	 						 hipMemcpyHostToDevice));
	gpuErrorCheck(hipMemcpy(d_params, params, amount_params * sizeof(double),
	 						 hipMemcpyHostToDevice));
	gpuErrorCheck(hipMemcpy(d_paramLinspaceA, linspaceA, size_A * sizeof(double),
	 						 hipMemcpyHostToDevice));
	gpuErrorCheck(hipMemcpy(d_paramLinspaceB, linspaceB, size_B * sizeof(double),
							 hipMemcpyHostToDevice));



	LLE_constants::calculateSystem<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(
	d_X,
	d_params,
    d_paramLinspaceA,
    d_paramLinspaceB,
    d_result
	);
    gpuErrorCheck(hipDeviceSynchronize());
    gpuErrorCheck(hipPeekAtLastError());


    printf("First calculation ended\n");

    double** h_result = new double*[size_A];
    for (int i = 0; i < size_A; ++i) {
        h_result[i] = new double[size_B];
        gpuErrorCheck(hipMemcpy(h_result[i], h_result_temp[i], size_B * sizeof(double), hipMemcpyDeviceToHost));
    }

    std::ofstream outFileStream(OUT_FILE_PATH);
    if (outFileStream.is_open()) {
        for (int i = 0; i < size_A; ++i) {
            for (int j = 0; j < size_B; ++j) {
                if (j > 0) outFileStream << ", ";
                outFileStream << (std::isnan(h_result[i][j]) ? 0 : h_result[i][j]);
            }
            outFileStream << "\n";
        }
        outFileStream.close();
    } else {
        std::cerr << "Output file open error: " << OUT_FILE_PATH << std::endl;
        exit(1);
    }

	delete[] linspaceA;
	delete[] linspaceB;
}

} //LLE_constants