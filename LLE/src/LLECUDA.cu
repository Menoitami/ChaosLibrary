#include "hip/hip_runtime.h"
#include <LLECUDA.cuh>
#include <string>
#include <iostream>
#include <fstream>
#include <hip/hiprtc.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_cooperative_groups.h>
namespace LLE_constants{
	
__device__ void calculateDiscreteModel(double *X, const double *a, const double h)
{
    X[0] += d_h1 * (-a[6] * X[1]);
    X[1] += d_h1 * (a[6] * X[0] + a[1] * X[2]);
    X[2] += d_h1 * (a[2] - a[3] * X[2] + a[4] * cos(a[5] * X[1])); // Убрали cos_term

    // Вычисление общего коэффициента для второй фазы
    double inv_den = 1.0 / (1.0 + a[3] * d_h2);

    // Обновления второй фазы
    X[2] = fma(d_h2, (a[2] + a[4] * cos(a[5] * X[1])), X[2]) * inv_den;
    X[1] += d_h2 * (a[6] * X[0] + a[1] * X[2]);
    X[0] += d_h2 * (-a[6] * X[1]);
}

__device__ bool loopCalculateDiscreteModel(double *x, const double *params,
                                                    const int amountOfIterations)
{
    for (int i = 0; i < amountOfIterations; ++i)
    {

        calculateDiscreteModel(x, params, d_h);
    }
    return true;
}


__global__ void calculateTransTime(
    double* X,
    double* params,
    const double* paramLinspaceA,
    const double* paramLinspaceB,
    double* semi_result
) {
    extern __shared__ double sh_mem[];

    const int idx_a = threadIdx.x + blockIdx.x * blockDim.x;
    const int idx_b = threadIdx.y + blockIdx.y * blockDim.y;
    if (idx_a >= d_size_linspace_A || idx_b >= d_size_linspace_B) return;

    const int thread_id = threadIdx.y * blockDim.x + threadIdx.x;
    const int total_size_per_thread = 2 * d_XSize + d_paramsSize;  // Полное состояние

    // Указатели в общей памяти
    double* my_sh_X = &sh_mem[thread_id * total_size_per_thread];
    double* my_sh_params = &sh_mem[thread_id * total_size_per_thread + d_XSize];
    double* my_sh_perturbated_X = &sh_mem[thread_id * total_size_per_thread + d_XSize + d_paramsSize];

    // Инициализация данных
    for (int i = 0; i < d_XSize; ++i) {
        my_sh_X[i] = X[i];
    }
    for (int i = 0; i < d_paramsSize; ++i) {
        my_sh_params[i] = params[i];
    }
    my_sh_params[d_idxParamA] = paramLinspaceA[idx_a];
    my_sh_params[d_idxParamB] = paramLinspaceB[idx_b];

    // Начальное вычисление модели
    loopCalculateDiscreteModel(my_sh_X, my_sh_params, d_amountOfTransPoints);

    // Генерация perturbated_X
    hiprandState_t state;
    hiprand_init(idx_a + idx_b * d_size_linspace_A, 0, 0, &state);

    double norm_factor = 0.0;
    for (int i = 0; i < d_XSize; ++i) {
        double z = hiprand_uniform(&state) - 0.5;
        norm_factor += z * z;
    }
    norm_factor = sqrt(norm_factor);

    for (int i = 0; i < d_XSize; ++i) {
        double z = (hiprand_uniform(&state) - 0.5) / norm_factor;
        my_sh_perturbated_X[i] = my_sh_X[i] + z * d_eps;
    }

    // Сохранение полного состояния в глобальную память
    double* res_sh_X = &semi_result[(idx_a * d_size_linspace_A + idx_b ) * total_size_per_thread];
    for (int i = 0; i < d_XSize; ++i) {
        res_sh_X[i] = my_sh_X[i];                    // X
        res_sh_X[i + d_XSize + d_paramsSize] = my_sh_perturbated_X[i];  // perturbated_X
    }
    for (int i = 0; i < d_paramsSize; ++i) {
        res_sh_X[i + d_XSize] = my_sh_params[i];     // params
    }
}

__global__ void calculateSystem(
    double* X,  // Оставлено для совместимости, не используется
    double* params,
    const double* paramLinspaceA,
    const double* paramLinspaceB,
    double* semi_result,
    double** result
) {
    extern __shared__ double sh_mem[];

    const int idx_a = threadIdx.x + blockIdx.x * blockDim.x;
    const int idx_b = threadIdx.y + blockIdx.y * blockDim.y;
    if (idx_a >= d_size_linspace_A || idx_b >= d_size_linspace_B) return;

    const int thread_id = threadIdx.y * blockDim.x + threadIdx.x;
    const int total_size_per_thread = 2 * d_XSize + d_paramsSize;

    // Указатели в общей памяти
    double* my_sh_X = &sh_mem[thread_id * total_size_per_thread];
    double* my_sh_params = &sh_mem[thread_id * total_size_per_thread + d_XSize];
    double* my_sh_perturbated_X = &sh_mem[thread_id * total_size_per_thread + d_XSize + d_paramsSize];

    // Загрузка полного состояния из глобальной памяти
    double* res_sh_X = &semi_result[(idx_a * d_size_linspace_A + idx_b ) * total_size_per_thread];
    for (int i = 0; i < d_XSize; ++i) {
        my_sh_X[i] = res_sh_X[i];                    // X
        my_sh_perturbated_X[i] = res_sh_X[i + d_XSize + d_paramsSize];  // perturbated_X
    }
    for (int i = 0; i < d_paramsSize; ++i) {
        my_sh_params[i] = res_sh_X[i + d_XSize];     // params
    }

    // Основной цикл вычислений
    double local_result = 0.0;
    const double inv_eps = 1.0 / d_eps;

    for (int i = 0; i <= d_amountOfCalcBlocks; ++i) {
        loopCalculateDiscreteModel(my_sh_X, my_sh_params, d_amountOfNTPoints);
        loopCalculateDiscreteModel(my_sh_perturbated_X, my_sh_params, d_amountOfNTPoints);

        // Расчет расстояния
        double distance = 0.0;
        for (int l = 0; l < d_XSize; ++l) {
            double diff = (my_sh_X[l] - my_sh_perturbated_X[l]) * inv_eps;
            distance += diff * diff;
        }
        distance = sqrt(distance);
        local_result += __logf(distance);

        // Обновление perturbated_X
        for (int j = 0; j < d_XSize; ++j) {
            my_sh_perturbated_X[j] = my_sh_X[j] - ((my_sh_X[j] - my_sh_perturbated_X[j]) / distance);
        }
    }

    // Запись результата
    atomicAdd(&result[idx_a][idx_b], local_result);
}

    
 


__host__ double* linspace(double start, double end, int num)
{
    // Allocate memory for num doubles
    double* result = new double[num];

    // Handle edge cases
    if (num < 0)
    {
        delete[] result;  // Clean up before throwing
        throw std::invalid_argument("received negative number of points");
    }
    if (num == 0)
    {
        return result;  // Return empty array
    }
    if (num == 1)
    {
        result[0] = start;  // Assign single value
        return result;
    }

    // Calculate step size
    double step = (end - start) / (num - 1);

    // Fill the array
    for (int i = 0; i < num; ++i)
    {
        result[i] = start + i * step;
    }

    return result;
}

__host__ void LLE2D(
	const double tMax,
	const double NT,
	const double h,
	const double eps,
	const double transientTime,
	const double* initialConditions,
	const int amount_init,
	const double* params,
	const int amount_params,
	const double* linspaceA_params,
	const double* linspaceB_params,
	const int* indicesOfMutVars,
	std::string		OUT_FILE_PATH)
{

	double* linspaceA = linspace(linspaceA_params[0], linspaceA_params[1], linspaceA_params[2]);
	double* linspaceB = linspace(linspaceB_params[0], linspaceB_params[1], linspaceB_params[2]);

	int amountOfNTPoints = static_cast<int>(NT / h);
	int amountOfTransPoints= static_cast<int>(transientTime / h);
	int amountOfAllPoints= static_cast<int>(tMax / h);

	size_t freeMemory;
	size_t totalMemory;

	gpuErrorCheck(hipMemGetInfo(&freeMemory, &totalMemory));

	const int size_A =  static_cast<int>(linspaceA_params[2]);
	const int size_B =  static_cast<int>(linspaceB_params[2]);
	int NT_steps = static_cast<int>(NT/h);

	int amount_of_calc_blocks = static_cast<int>(amountOfAllPoints/amountOfNTPoints) + 1;

	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_idxParamA), &indicesOfMutVars[0], sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_idxParamB), &indicesOfMutVars[1], sizeof(int)));

	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_size_linspace_A), &size_A, sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_size_linspace_B), &size_B, sizeof(int)));

	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_h), &h, sizeof(double)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_transTime), &transientTime, sizeof(double)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_Nt_steps), &NT_steps, sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_paramsSize), &amount_params, sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_XSize), &amount_init, sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_amountOfNTPoints), &amountOfNTPoints, sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_amountOfTransPoints), &amountOfTransPoints, sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_amountOfAllpoints), &amountOfAllPoints, sizeof(int)));
    gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_amountOfCalcBlocks), &amount_of_calc_blocks, sizeof(int)));

    double h_h1 = params[0] * h;
    double h_h2 = (1 - params[0]) * h;
    gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_h1), &h_h1, sizeof(double)));
    gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_h2), &h_h2, sizeof(double)));
    gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_eps), &eps, sizeof(double)));


    int max_threads_y =  16;  
    int max_threads_x =  16;  

    int gridSizeY = (size_B - 1) / max_threads_y;
    int gridSizeX =  (size_A - 1) / max_threads_x;


    // Define thread block and grid dimensions
    dim3 threadsPerBlock(max_threads_x, max_threads_y, 1);  // e.g., (16, 8, 1)
    dim3 blocksPerGrid(gridSizeX, gridSizeY, 1);

    size_t sharedMemSizeTrans = (max_threads_x * max_threads_y) * (amount_init + amount_params) * sizeof(double);
    size_t sharedMemSize = (max_threads_x * max_threads_y) * (2 * amount_init + amount_params) * sizeof(double);  // For thread pairs
    printf("Total shared memory: %zu bytes\n", sharedMemSize);


    double** d_result;
    double** h_result_temp = new double*[size_A];
    gpuErrorCheck(hipMalloc(&d_result, size_A * sizeof(double*)));
    for (int i = 0; i < size_A; ++i) {
        gpuErrorCheck(hipMalloc(&h_result_temp[i], size_B * sizeof(double)));
        double zero = 0.0;

        for (int j = 0; j < size_B; ++j) {
            gpuErrorCheck(hipMemcpy(h_result_temp[i] + j, &zero, sizeof(double), hipMemcpyHostToDevice));
        }
    }
    gpuErrorCheck(hipMemcpy(d_result, h_result_temp, size_A * sizeof(double*), hipMemcpyHostToDevice));


    double* d_semi_result;
    gpuErrorCheck(hipMalloc(&d_semi_result, size_A * size_B * (2 * amount_init + amount_params) * sizeof(double)));

    double* d_paramLinspaceA;
    double* d_paramLinspaceB;
    double* d_X;
    double* d_params;

    gpuErrorCheck(hipMalloc(&d_X, amount_init * sizeof(double)));
    gpuErrorCheck(hipMalloc(&d_params, amount_params * sizeof(double)));
    gpuErrorCheck(hipMalloc(&d_paramLinspaceA, size_A * sizeof(double)));
    gpuErrorCheck(hipMalloc(&d_paramLinspaceB, size_B * sizeof(double)));

    gpuErrorCheck(hipMemcpy(d_X, initialConditions, amount_init * sizeof(double), hipMemcpyHostToDevice));
    gpuErrorCheck(hipMemcpy(d_params, params, amount_params * sizeof(double), hipMemcpyHostToDevice));
    gpuErrorCheck(hipMemcpy(d_paramLinspaceA, linspaceA, size_A * sizeof(double), hipMemcpyHostToDevice));
    gpuErrorCheck(hipMemcpy(d_paramLinspaceB, linspaceB, size_B * sizeof(double), hipMemcpyHostToDevice));

    // Первый вызов: расчет trans_time и perturbated_X
    LLE_constants::calculateTransTime<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(
        d_X,
        d_params,
        d_paramLinspaceA,
        d_paramLinspaceB,
        d_semi_result
    );
    gpuErrorCheck(hipDeviceSynchronize());
    gpuErrorCheck(hipPeekAtLastError());

    // Второй вызов: расчет системы
    LLE_constants::calculateSystem<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(
        d_X,
        d_params,
        d_paramLinspaceA,
        d_paramLinspaceB,
        d_semi_result,
        d_result
    );
    gpuErrorCheck(hipDeviceSynchronize());
    gpuErrorCheck(hipPeekAtLastError());

    printf("Calculations ended\n");

    double** h_result = new double*[size_A];
    for (int i = 0; i < size_A; ++i) {
        h_result[i] = new double[size_B];
        gpuErrorCheck(hipMemcpy(h_result[i], h_result_temp[i], size_B * sizeof(double), hipMemcpyDeviceToHost));
    }

    std::ofstream outFileStream(OUT_FILE_PATH);
    if (outFileStream.is_open()) {
        for (int i = 0; i < size_A; ++i) {
            for (int j = 0; j < size_B; ++j) {
                if (j > 0) outFileStream << ", ";
                outFileStream << (std::isnan(h_result[i][j]) ? 0 : h_result[i][j]);
            }
            outFileStream << "\n";
        }
        outFileStream.close();
    } else {
        std::cerr << "Output file open error: " << OUT_FILE_PATH << std::endl;
        exit(1);
    }

	delete[] linspaceA;
	delete[] linspaceB;
}

} //LLE_constants