#include "hip/hip_runtime.h"
#include <LLECUDA.cuh>
#include <string>
#include <iostream>
#include <fstream>
#include <hip/hiprtc.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_cooperative_groups.h>
namespace LLE_constants{
	


__device__ void loopCalculateDiscreteModel(double *X, const double *a,
                                                    const int amountOfIterations)
{

    double x0 = X[0];
    double x1 = X[1];
    double x2 = X[2];

    // Загружаем элементы массива a в регистры
    double a0 = a[0]; // Предполагаем, что a[0] может использоваться где-то вне видимого кода
    double a1 = a[1];
    double a2 = a[2];
    double a3 = a[3];
    double a4 = a[4];
    double a5 = a[5];
    double a6 = a[6];

    #pragma unroll
    for (int i = 0; i < amountOfIterations; ++i)
    {
        double cos_term = cos(a5 * x1);
        x0 = __fma_rn(d_h1, (-a6 * x1), x0);          // x0 += d_h1 * (-a6 * x1)
        x1 = __fma_rn(d_h1, (a6 * x0 + a1 * x2), x1); // x1 += d_h1 * (a6 * x0 + a1 * x2)
        x2 = __fma_rn(d_h1, (a2 - a3 * x2 + a4 * cos_term), x2); // x2 += d_h1 * (a2 - a3 * x2 + a4 * cos_term)

        // Вычисление общего коэффициента для второй фазы
        float inv_den = __frcp_rn(__fmaf_rn(a3, d_h2, 1.0f));     // Здесь fused не нужен, так как нет умножения с последующим сложением

        // Вторая фаза
        x2 = __fma_rn(d_h2, (a2 + a4 * cos_term), x2) * inv_den; // x2 = fma(d_h2, (a2 + a4 * cos_term), x2) * inv_den
        x1 = __fma_rn(d_h2, (a6 * x0 + a1 * x2), x1); // x1 += d_h2 * (a6 * x0 + a1 * x2)
        x0 = __fma_rn(d_h2, (-a6 * x1), x0);          // x0 += d_h2 * (-a6 * x1)
    }

    X[0] = x0;
    X[1] = x1;
    X[2] = x2;
}


__global__ void calculateTransTime(
    double* X,
    double* params,
    const double* paramLinspaceA,
    const double* paramLinspaceB,
    double* semi_result
) {
    extern __shared__ double sh_mem[];

    const int idx_a = threadIdx.x + blockIdx.x * blockDim.x;
    const int idx_b = threadIdx.y + blockIdx.y * blockDim.y;
    if (idx_a >= d_size_linspace_A || idx_b >= d_size_linspace_B) return;

    const int thread_id = threadIdx.y * blockDim.x + threadIdx.x;
    const int total_size_per_thread = 2 * d_XSize + d_paramsSize;  // Полное состояние

    // Указатели в общей памяти
    double* my_sh_X = &sh_mem[thread_id * total_size_per_thread];
    double* my_sh_params = &sh_mem[thread_id * total_size_per_thread + d_XSize];
    double* my_sh_perturbated_X = &sh_mem[thread_id * total_size_per_thread + d_XSize + d_paramsSize];

    // Инициализация данных
    for (int i = 0; i < d_XSize; ++i) {
        my_sh_X[i] = X[i];
    }
    for (int i = 0; i < d_paramsSize; ++i) {
        my_sh_params[i] = params[i];
    }
    my_sh_params[d_idxParamA] = paramLinspaceA[idx_a];
    my_sh_params[d_idxParamB] = paramLinspaceB[idx_b];

    // Начальное вычисление модели
    loopCalculateDiscreteModel(my_sh_X, my_sh_params, d_amountOfTransPoints);

    // Генерация perturbated_X
    hiprandState_t state;
    hiprand_init(idx_a + idx_b * d_size_linspace_A, 0, 0, &state);

    float norm_factor = 0.0f;
    for (int i = 0; i < d_XSize; ++i) {
        float z = hiprand_uniform(&state) - 0.5f;
        norm_factor = __fmaf_rn(z, z, norm_factor);
    }
    norm_factor = __fmul_rn(rsqrtf(norm_factor), norm_factor);
    for (int i = 0; i < d_XSize; ++i) {
        float z = __fdiv_rn(hiprand_uniform(&state) - 0.5f, norm_factor);
        my_sh_perturbated_X[i] = __fmaf_rn(z, (float)d_eps, my_sh_X[i]);
    }

    // Сохранение полного состояния в глобальную память
    double* res_sh_X = &semi_result[(idx_a * d_size_linspace_A + idx_b ) * total_size_per_thread];
    for (int i = 0; i < d_XSize; ++i) {
        res_sh_X[i] = my_sh_X[i];                    // X
        res_sh_X[i + d_XSize + d_paramsSize] = my_sh_perturbated_X[i];  // perturbated_X
    }
    for (int i = 0; i < d_paramsSize; ++i) {
        res_sh_X[i + d_XSize] = my_sh_params[i];     // params
    }
}

__global__ void calculateSystem(
    double* X,  // Оставлено для совместимости, не используется
    double* params,
    const double* paramLinspaceA,
    const double* paramLinspaceB,
    double* semi_result,
    double** result
) {
    extern __shared__ double sh_mem[];

    const int idx_a = threadIdx.x + blockIdx.x * blockDim.x;
    const int idx_b = threadIdx.y + blockIdx.y * blockDim.y;
    if (idx_a >= d_size_linspace_A || idx_b >= d_size_linspace_B) return;

    const int thread_id = threadIdx.y * blockDim.x + threadIdx.x;
    const int total_size_per_thread = 2 * d_XSize + d_paramsSize;

    // Указатели в общей памяти
    double* my_sh_X = &sh_mem[thread_id * total_size_per_thread];
    double* my_sh_params = &sh_mem[thread_id * total_size_per_thread + d_XSize];
    double* my_sh_perturbated_X = &sh_mem[thread_id * total_size_per_thread + d_XSize + d_paramsSize];

    // Загрузка полного состояния из глобальной памяти
    double* res_sh_X = &semi_result[(idx_a * d_size_linspace_A + idx_b ) * total_size_per_thread];
    for (int i = 0; i < d_XSize; ++i) {
        my_sh_X[i] = res_sh_X[i];                    // X
        my_sh_perturbated_X[i] = res_sh_X[i + d_XSize + d_paramsSize];  // perturbated_X
    }
    for (int i = 0; i < d_paramsSize; ++i) {
        my_sh_params[i] = res_sh_X[i + d_XSize];     // params
    }

    // Основной цикл вычислений
    double local_result = 0.0;
    const double inv_eps = 1.0 / d_eps;

    for (int i = 0; i <= d_amountOfCalcBlocks; ++i) {
        loopCalculateDiscreteModel(my_sh_X, my_sh_params, d_amountOfNTPoints);
        loopCalculateDiscreteModel(my_sh_perturbated_X, my_sh_params, d_amountOfNTPoints);

        // Расчет расстояния
        double distance = 0.0;
        for (int l = 0; l < d_XSize; ++l) {
            double diff = (my_sh_X[l] - my_sh_perturbated_X[l]) * inv_eps;
            distance += diff * diff;
        }
        distance = sqrt(distance);
        local_result += __logf(distance);

        // Обновление perturbated_X
        for (int j = 0; j < d_XSize; ++j) {
            my_sh_perturbated_X[j] = my_sh_X[j] - ((my_sh_X[j] - my_sh_perturbated_X[j]) / distance);
        }
    }

    // Запись результата
    atomicAdd(&result[idx_a][idx_b], local_result);
}

    
 


__host__ double* linspace(double start, double end, int num)
{
    // Allocate memory for num doubles
    double* result = new double[num];

    // Handle edge cases
    if (num < 0)
    {
        delete[] result;  // Clean up before throwing
        throw std::invalid_argument("received negative number of points");
    }
    if (num == 0)
    {
        return result;  // Return empty array
    }
    if (num == 1)
    {
        result[0] = start;  // Assign single value
        return result;
    }

    // Calculate step size
    double step = (end - start) / (num - 1);

    // Fill the array
    for (int i = 0; i < num; ++i)
    {
        result[i] = start + i * step;
    }

    return result;
}

__host__ void LLE2D(
	const double tMax,
	const double NT,
	const double h,
	const double eps,
	const double transientTime,
	const double* initialConditions,
	const int amount_init,
	const double* params,
	const int amount_params,
	const double* linspaceA_params,
	const double* linspaceB_params,
	const int* indicesOfMutVars,
	std::string		OUT_FILE_PATH)
{

	double* linspaceA = linspace(linspaceA_params[0], linspaceA_params[1], linspaceA_params[2]);
	double* linspaceB = linspace(linspaceB_params[0], linspaceB_params[1], linspaceB_params[2]);

	int amountOfNTPoints = static_cast<int>(NT / h);
	int amountOfTransPoints= static_cast<int>(transientTime / h);
	int amountOfAllPoints= static_cast<int>(tMax / h);

	size_t freeMemory;
	size_t totalMemory;

	gpuErrorCheck(hipMemGetInfo(&freeMemory, &totalMemory));

	const int size_A =  static_cast<int>(linspaceA_params[2]);
	const int size_B =  static_cast<int>(linspaceB_params[2]);
	int NT_steps = static_cast<int>(NT/h);

	int amount_of_calc_blocks = static_cast<int>(amountOfAllPoints/amountOfNTPoints) + 1;

	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_idxParamA), &indicesOfMutVars[0], sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_idxParamB), &indicesOfMutVars[1], sizeof(int)));

	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_size_linspace_A), &size_A, sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_size_linspace_B), &size_B, sizeof(int)));

	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_h), &h, sizeof(double)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_transTime), &transientTime, sizeof(double)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_Nt_steps), &NT_steps, sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_paramsSize), &amount_params, sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_XSize), &amount_init, sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_amountOfNTPoints), &amountOfNTPoints, sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_amountOfTransPoints), &amountOfTransPoints, sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_amountOfAllpoints), &amountOfAllPoints, sizeof(int)));
    gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_amountOfCalcBlocks), &amount_of_calc_blocks, sizeof(int)));

    double h_h1 = params[0] * h;
    double h_h2 = (1 - params[0]) * h;
    gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_h1), &h_h1, sizeof(double)));
    gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_h2), &h_h2, sizeof(double)));
    gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_eps), &eps, sizeof(double)));


    int max_threads_y =  16;  
    int max_threads_x =  16;  

    int gridSizeY = (size_B+max_threads_y-1) / max_threads_y;
    int gridSizeX =  (size_A+max_threads_x-1) / max_threads_x;


    // Define thread block and grid dimensions
    dim3 threadsPerBlock(max_threads_x, max_threads_y, 1); 
    dim3 blocksPerGrid(gridSizeX, gridSizeY, 1);

    size_t sharedMemSizeTrans = (max_threads_x * max_threads_y) * (amount_init + amount_init + amount_params) * sizeof(double);
    size_t sharedMemSize = (max_threads_x * max_threads_y) * (2 * amount_init + amount_params) * sizeof(double);  // For thread pairs
    printf("Total shared memory: %zu bytes\n", sharedMemSize);


    double** d_result;
    double** h_result_temp = new double*[size_A];
    gpuErrorCheck(hipMalloc(&d_result, size_A * sizeof(double*)));
    for (int i = 0; i < size_A; ++i) {
        gpuErrorCheck(hipMalloc(&h_result_temp[i], size_B * sizeof(double)));
        double zero = 0.0;

        for (int j = 0; j < size_B; ++j) {
            gpuErrorCheck(hipMemcpy(h_result_temp[i] + j, &zero, sizeof(double), hipMemcpyHostToDevice));
        }
    }
    gpuErrorCheck(hipMemcpy(d_result, h_result_temp, size_A * sizeof(double*), hipMemcpyHostToDevice));


    double* d_semi_result;
    gpuErrorCheck(hipMalloc(&d_semi_result, size_A * size_B * (2 * amount_init + amount_params) * sizeof(double)));

    double* d_paramLinspaceA;
    double* d_paramLinspaceB;
    double* d_X;
    double* d_params;

    gpuErrorCheck(hipMalloc(&d_X, amount_init * sizeof(double)));
    gpuErrorCheck(hipMalloc(&d_params, amount_params * sizeof(double)));
    gpuErrorCheck(hipMalloc(&d_paramLinspaceA, size_A * sizeof(double)));
    gpuErrorCheck(hipMalloc(&d_paramLinspaceB, size_B * sizeof(double)));

    gpuErrorCheck(hipMemcpy(d_X, initialConditions, amount_init * sizeof(double), hipMemcpyHostToDevice));
    gpuErrorCheck(hipMemcpy(d_params, params, amount_params * sizeof(double), hipMemcpyHostToDevice));
    gpuErrorCheck(hipMemcpy(d_paramLinspaceA, linspaceA, size_A * sizeof(double), hipMemcpyHostToDevice));
    gpuErrorCheck(hipMemcpy(d_paramLinspaceB, linspaceB, size_B * sizeof(double), hipMemcpyHostToDevice));

    // Первый вызов: расчет trans_time и perturbated_X
    LLE_constants::calculateTransTime<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(
        d_X,
        d_params,
        d_paramLinspaceA,
        d_paramLinspaceB,
        d_semi_result
    );
    gpuErrorCheck(hipDeviceSynchronize());
    gpuErrorCheck(hipPeekAtLastError());

    // Второй вызов: расчет системы
    LLE_constants::calculateSystem<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(
        d_X,
        d_params,
        d_paramLinspaceA,
        d_paramLinspaceB,
        d_semi_result,
        d_result
    );
    gpuErrorCheck(hipDeviceSynchronize());
    gpuErrorCheck(hipPeekAtLastError());

    printf("Calculations ended\n");

    double** h_result = new double*[size_A];
    for (int i = 0; i < size_A; ++i) {
        h_result[i] = new double[size_B];
        gpuErrorCheck(hipMemcpy(h_result[i], h_result_temp[i], size_B * sizeof(double), hipMemcpyDeviceToHost));
    }

    std::ofstream outFileStream(OUT_FILE_PATH);
    if (outFileStream.is_open()) {
        for (int i = 0; i < size_A; ++i) {
            for (int j = 0; j < size_B; ++j) {
                if (j > 0) outFileStream << ", ";
                outFileStream << (std::isnan(h_result[i][j]) ? 0 : h_result[i][j]);
            }
            outFileStream << "\n";
        }
        outFileStream.close();
    } else {
        std::cerr << "Output file open error: " << OUT_FILE_PATH << std::endl;
        exit(1);
    }

	delete[] linspaceA;
	delete[] linspaceB;
}

} //LLE_constants