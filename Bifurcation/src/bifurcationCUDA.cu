#include "hip/hip_runtime.h"
#include <bifurcationCUDA.cuh>
#include <string>
#include <iostream>
#include <fstream>
#include <iomanip>

namespace Bifurcation_constants {
__constant__ double d_tMax;
__constant__ int d_nPts;
__constant__ double d_h;
__constant__ int d_amountOfInitialConditions;

__constant__ int d_writableVar;
__constant__ double d_maxValue;
__constant__ double d_transientTime;

__constant__ int d_amountOfValues;
__constant__ int d_preScaller;
__constant__ double d_eps;


__constant__ int d_sizeOfBlock;
__constant__ int d_dimension;
__constant__ int d_amountOfIterations;


__constant__ int d_nPtsLimiter;

__constant__ int d_amountOfPointsInBlock;
__constant__ int d_amountOfPointsForSkip;
__constant__ int d_originalNPtsLimiter;

__constant__ int d_amountOfCalculatedPoints;


__host__ void bifurcation2D(
	const double	tMax,								// Время моделирования системы
	const int		nPts,								// Разрешение диаграммы
	const double	h,									// Шаг интегрирования
	const int		amountOfInitialConditions,			// Количество начальных условий ( уравнений в системе )
	const double* initialConditions,					// Массив с начальными условиями
	const double* ranges,								// Диапазоны изменения параметров
	const int* indicesOfMutVars,					// Индексы изменяемых параметров
	const int		writableVar,						// Индекс уравнения, по которому будем строить диаграмму
	const double	maxValue,							// Максимальное значение (по модулю), выше которого система считаемся "расшедшейся"
	const double	transientTime,						// Время, которое будет промоделировано перед расчетом диаграммы
	const double* values,								// Параметры
	const int		amountOfValues,						// Количество параметров
	const int		preScaller,							// Множитель, который уменьшает время и объем расчетов (будет рассчитываться только каждая 'preScaller' точка)
	const double	eps,
	std::string		OUT_FILE_PATH)								// Эпсилон для алгоритма DBSCAN 
{
	int amountOfPointsInBlock = tMax / h / preScaller;

	int amountOfPointsForSkip = transientTime / h;

	size_t freeMemory;											
	size_t totalMemory;											
	gpuErrorCheck(hipMemGetInfo(&freeMemory, &totalMemory));	
	freeMemory *= 0.95;				
	size_t nPtsLimiter = freeMemory / (sizeof(double) * amountOfPointsInBlock * 3);
	nPtsLimiter = nPtsLimiter > (nPts * nPts) ? (nPts * nPts) : nPtsLimiter;
	size_t originalNPtsLimiter = nPtsLimiter;				

	int* h_dbscanResult = new int[nPtsLimiter * sizeof(int)];



	double* d_data;					// Указатель на массив в памяти GPU для хранения траектории системы
	double* d_ranges;				// Указатель на массив с диапазоном изменения переменной
	int* d_indicesOfMutVars;		// Указатель на массив с индексом изменяемой переменной в массиве values
	double* d_initialConditions;	// Указатель на массив с начальными условиями
	double* d_values;				// Указатель на массив с параметрами

	int* d_amountOfPeaks;		// Указатель на массив в GPU с кол-вом пиков в каждой системе.
	double* d_intervals;			// Указатель на массив в GPU с межпиковыми интервалами пиков
	int* d_dbscanResult;			// Указатель на массив в GPU результирующей матрицы (диаграммы) в GPU
	double* d_helpfulArray;			// Указатель на массив в GPU на вспомогательный массив


	gpuErrorCheck(hipMalloc((void**)& d_data, nPtsLimiter * amountOfPointsInBlock * sizeof(double)));
	gpuErrorCheck(hipMalloc((void**)& d_ranges, 4 * sizeof(double)));
	gpuErrorCheck(hipMalloc((void**)& d_indicesOfMutVars, 2 * sizeof(int)));
	gpuErrorCheck(hipMalloc((void**)& d_initialConditions, amountOfInitialConditions * sizeof(double)));
	gpuErrorCheck(hipMalloc((void**)& d_values, amountOfValues * sizeof(double)));

	gpuErrorCheck(hipMalloc((void**)& d_amountOfPeaks, nPtsLimiter * sizeof(int)));
	gpuErrorCheck(hipMalloc((void**)& d_intervals, nPtsLimiter * amountOfPointsInBlock * sizeof(double)));
	gpuErrorCheck(hipMalloc((void**)& d_dbscanResult, nPtsLimiter * sizeof(int)));
	gpuErrorCheck(hipMalloc((void**)& d_helpfulArray, nPtsLimiter * amountOfPointsInBlock * sizeof(double)));

	gpuErrorCheck(hipMemcpy(d_ranges, ranges, 4 * sizeof(double), hipMemcpyKind::hipMemcpyHostToDevice));
	gpuErrorCheck(hipMemcpy(d_indicesOfMutVars, indicesOfMutVars, 2 * sizeof(int), hipMemcpyKind::hipMemcpyHostToDevice));
	gpuErrorCheck(hipMemcpy(d_initialConditions, initialConditions, amountOfInitialConditions * sizeof(double), hipMemcpyKind::hipMemcpyHostToDevice));
	gpuErrorCheck(hipMemcpy(d_values, values, amountOfValues * sizeof(double), hipMemcpyKind::hipMemcpyHostToDevice));

	size_t amountOfIteration = (size_t)ceil((double)(nPts * nPts) / (double)nPtsLimiter);


	std::ofstream outFileStream;
	outFileStream.open(OUT_FILE_PATH);

	int stringCounter = 0; 


	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_tMax), &tMax, sizeof(double)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_nPts), &nPts, sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_h), &h, sizeof(double)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_amountOfInitialConditions), &amountOfInitialConditions, sizeof(int)));

	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_writableVar), &writableVar, sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_maxValue), &maxValue, sizeof(double)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_transientTime), &transientTime, sizeof(double)));

	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_amountOfValues), &amountOfValues, sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_preScaller), &preScaller, sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_eps), &eps, sizeof(double)));
	
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_amountOfPointsInBlock), &amountOfPointsInBlock, sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_originalNPtsLimiter), &originalNPtsLimiter, sizeof(int)));
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_amountOfPointsForSkip), &amountOfPointsForSkip, sizeof(int)));

	int dimension = 2;
	gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_dimension), &dimension, sizeof(int)));


	for (int i = 0; i < amountOfIteration; ++i)
	{

		if (i == amountOfIteration - 1)
			nPtsLimiter = (nPts * nPts) - (nPtsLimiter * i);

		int blockSize;			// Переменная для хранения размера блока
		int minGridSize;		// Переменная для хранения минимального размера сетки
		int gridSize;			// Переменная для хранения сетки

		blockSize = 20000 / ((amountOfInitialConditions + amountOfValues) * sizeof(double));

		gridSize = (nPtsLimiter + blockSize - 1) / blockSize;

		int calculatedPoints = i * originalNPtsLimiter;
		gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_nPtsLimiter), &nPtsLimiter, sizeof(int)));
		gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_amountOfCalculatedPoints), &calculatedPoints, sizeof(int)));

		calculateDiscreteModelCUDA << <gridSize, blockSize, (amountOfInitialConditions + amountOfValues) * sizeof(double) * blockSize >> >(
						d_ranges,
						d_indicesOfMutVars,
						d_initialConditions,
						d_values,
						d_data,
						d_amountOfPeaks);

		gpuGlobalErrorCheck();

		gpuErrorCheck(hipDeviceSynchronize());

		hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, peakFinderCUDA, 0, nPtsLimiter);
		gridSize = (nPtsLimiter + blockSize - 1) / blockSize;

		peakFinderCUDA << <gridSize, blockSize >> >
			(	d_data,					
				d_amountOfPeaks,			
				d_data,						
				d_intervals,				
				h * (double)preScaller);	

		gpuGlobalErrorCheck();

		gpuErrorCheck(hipDeviceSynchronize());

		hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, dbscanCUDA, 0, nPtsLimiter);
		gridSize = (nPtsLimiter + blockSize - 1) / blockSize;

		dbscanCUDA << <gridSize, blockSize >> > (	
				d_data,
				amountOfPointsInBlock,
				nPtsLimiter,
				d_amountOfPeaks,
				d_intervals,
				d_helpfulArray,
				eps,
				d_dbscanResult);

		gpuGlobalErrorCheck();
		gpuErrorCheck(hipDeviceSynchronize());

		gpuErrorCheck(hipMemcpy(h_dbscanResult, d_dbscanResult, nPtsLimiter * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost));

		outFileStream << std::setprecision(12);

		for (size_t i = 0; i < nPtsLimiter; ++i)
			if (outFileStream.is_open())
			{
				if (stringCounter != 0)
					outFileStream << ", ";
				if (stringCounter == nPts)
				{
					outFileStream << "\n";
					stringCounter = 0;
				}
				outFileStream << h_dbscanResult[i];
				++stringCounter;
			}
			else
			{
				exit(1);
			}
	}


	gpuErrorCheck(hipFree(d_data));
	gpuErrorCheck(hipFree(d_ranges));
	gpuErrorCheck(hipFree(d_indicesOfMutVars));
	gpuErrorCheck(hipFree(d_initialConditions));
	gpuErrorCheck(hipFree(d_values));

	gpuErrorCheck(hipFree(d_amountOfPeaks));
	gpuErrorCheck(hipFree(d_intervals));
	gpuErrorCheck(hipFree(d_dbscanResult));
	gpuErrorCheck(hipFree(d_helpfulArray));

	delete[] h_dbscanResult;
}



__global__ void calculateDiscreteModelCUDA(
	double*			ranges, 
	int*			indicesOfMutVars, 
	double*			initialConditions,
	const double*	values, 
	double*			data, 
	int*			maxValueCheckerArray)
{
	extern __shared__ double s[];

	double* localX = s + ( threadIdx.x * d_amountOfInitialConditions );
	double* localValues = s + ( blockDim.x * d_amountOfInitialConditions ) + ( threadIdx.x * d_amountOfValues );

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= d_nPtsLimiter)		
			return;

	for ( int i = 0; i < d_amountOfInitialConditions; ++i )
		localX[i] = initialConditions[i];

	for (int i = 0; i < d_amountOfValues; ++i)
		localValues[i] = values[i];

	for (int i = 0; i < d_dimension; ++i)
		localValues[indicesOfMutVars[i]] = getValueByIdx(d_amountOfCalculatedPoints + idx, 
			d_nPts, ranges[i * 2], ranges[i * 2 + 1], i);

	int flag = loopCalculateDiscreteModel_int(localX, localValues, d_h, d_amountOfPointsForSkip,
		d_amountOfInitialConditions, d_writableVar, d_maxValue, nullptr, idx * d_amountOfPointsInBlock);

	if (flag == 1)
		flag = loopCalculateDiscreteModel_int(localX, localValues, d_h, d_amountOfPointsInBlock,
			d_amountOfInitialConditions, d_writableVar, d_maxValue, data, idx * d_amountOfPointsInBlock);


	if (maxValueCheckerArray != nullptr) {
		maxValueCheckerArray[idx] = flag;
	}


	return;
}

__global__ void peakFinderCUDA(
    double* data, 
    int* amountOfPeaks, 
    double* outPeaks, 
    double* timeOfPeaks, 
    double h)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= d_nPtsLimiter)
		return;

	if (amountOfPeaks[idx] == -1) {
		amountOfPeaks[idx] = -1;
		return;
	}

	if (amountOfPeaks[idx] == 0) {
		amountOfPeaks[idx] = 0;
		return;
	}

	amountOfPeaks[idx] = peakFinder(data, idx * d_amountOfPointsInBlock, d_amountOfPointsInBlock, outPeaks, timeOfPeaks, h);
	return;
}

__global__ void dbscanCUDA(
    double* data, 
    const int sizeOfBlock, 
    const int amountOfBlocks,
    const int* amountOfPeaks, 
    double* intervals, 
    double* helpfulArray,
    const double eps, 
    int* outData)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= amountOfBlocks)
		return;

	if (amountOfPeaks[idx] == -1) {
		outData[idx] = -1;
		return;
	}

	if (amountOfPeaks[idx] == 0) {
		outData[idx] = 0;
		return;
	}

	outData[idx] = dbscan(data, intervals, helpfulArray, idx * sizeOfBlock, amountOfPeaks[idx], sizeOfBlock, idx, eps, outData);
}

__device__ double getValueByIdx(
    const int idx, 
    const int nPts,
    const double startRange, 
    const double finishRange, 
    const int valueNumber)
{
    // Предварительно вычисляем степень
    double divisor;
    switch(valueNumber) {
        case 0: divisor = 1.0; break;
        case 1: divisor = nPts; break;
        case 2: divisor = nPts * nPts; break;
        default: divisor = __powf(nPts, valueNumber);
    }
    
    int normalizedIdx = (idx / (int)divisor) % nPts;
    double scale = (finishRange - startRange) / (nPts - 1);
    return startRange + normalizedIdx * scale;
}

__device__ int loopCalculateDiscreteModel_int(
    double* x, 
    const double* values,
    const double h, 
    const int amountOfIterations, 
    const int amountOfX, 
    int writableVar, 
    const double maxValue, 
    double* data,
    const int startDataIndex)
{
	const int MAX_REG_SIZE = 12;
	double xPrev[MAX_REG_SIZE];

	for (int i = 0; i < amountOfIterations; ++i) {
		for (int j = 0; j < amountOfX; ++j) {
			xPrev[j] = x[j];
		}

		if (data != nullptr) 
			data[startDataIndex + i] = (x[writableVar]);

		calculateDiscreteModel(x, values, h);

		double val = x[writableVar];
		if (val != val || val == val + 1.0) {
			return 0;
		}

		if (maxValue != 0)
			if (fabsf(x[writableVar]) > maxValue) {
				return 0;
			}
	}

	double tempResult = 0;

	for (int j = 0; j < amountOfX; ++j) {
		tempResult += ((x[j] - xPrev[j]) * (x[j] - xPrev[j]));
	}

	if (sqrtf(fabsf(tempResult)) < 1e-9) {
		return -1;
	}

	return 1;
}

__device__ int peakFinder(double* data, const int startDataIndex, 
	const int amountOfPoints, double* outPeaks, double* timeOfPeaks, double h)
{
	int amountOfPeaks = 0;
	
	for ( int i = startDataIndex + 1; i < startDataIndex + amountOfPoints - 1; ++i )
	{
		if ( data[i] - data[i - 1] > 1e-13 && data[i] >= data[i + 1] )
		{
			for ( int j = i; j < startDataIndex + amountOfPoints - 1; ++j )
			{
				if ( data[j] < data[j + 1] )
				{
					i = j + 1;
					break;
				}
				if ( data[j] - data[j + 1] > 1e-13  )
				{
					if ( outPeaks != nullptr )
						outPeaks[startDataIndex + amountOfPeaks] = data[j];
					if ( timeOfPeaks != nullptr )
						timeOfPeaks[startDataIndex + amountOfPeaks] = trunc( ( (double)j + (double)i ) / (double)2 );
					++amountOfPeaks;
					i = j + 1;
					break;
				}
			}
		}
	}
	if ( amountOfPeaks > 1 ) {
		for ( size_t i = 0; i < amountOfPeaks - 1; i++ )
		{
			if ( outPeaks != nullptr )
				outPeaks[startDataIndex + i] = outPeaks[startDataIndex + i + 1];
			if ( timeOfPeaks != nullptr )
				timeOfPeaks[startDataIndex + i] = (double)( timeOfPeaks[startDataIndex + i + 1] - timeOfPeaks[startDataIndex + i] ) * h;
		}
		amountOfPeaks = amountOfPeaks - 1;
	}
	else {
		amountOfPeaks = 0;
	}


	return amountOfPeaks;
}

__device__ int dbscan(double* data, double* intervals, double* helpfulArray, 
	const int startDataIndex, const int amountOfPeaks, const int sizeOfHelpfulArray,
	const int idx, const double eps, int* outData)
{

	if (amountOfPeaks <= 0)
		return 0;

	if (amountOfPeaks == 1)
		return 1;

	int cluster = 0;
	int NumNeibor = 0;

	for (int i = startDataIndex; i < startDataIndex + sizeOfHelpfulArray; ++i) {
		helpfulArray[i] = 0;
	}

	for (int i = 0; i < amountOfPeaks; i++) {
		data[startDataIndex + i] = 0; 
	}

	for (int i = 0; i < amountOfPeaks; i++)
		if (NumNeibor >= 1)
		{
			i = helpfulArray[startDataIndex + amountOfPeaks + NumNeibor - 1];
			helpfulArray[startDataIndex + amountOfPeaks + NumNeibor - 1] = 0;
			NumNeibor = NumNeibor - 1;
			for (int k = 0; k < amountOfPeaks - 1; k++) {
				if (i != k && helpfulArray[startDataIndex + k] == 0) {
					if (distance(data[startDataIndex + i], intervals[startDataIndex + i], data[startDataIndex + k], intervals[startDataIndex + k]) < eps) {
						helpfulArray[startDataIndex + k] = cluster;
						helpfulArray[startDataIndex + amountOfPeaks + k] = k;
						++NumNeibor;
					}
				}
				
			}
		}
		else if (helpfulArray[startDataIndex + i] == 0) {
			NumNeibor = 0;
			++cluster;
			helpfulArray[startDataIndex + i] = cluster;
			for (int k = 0; k < amountOfPeaks - 1; k++) {
				if (i != k && helpfulArray[startDataIndex + k] == 0) {
					if (distance(data[startDataIndex + i], intervals[startDataIndex + i], data[startDataIndex + k], intervals[startDataIndex + k]) < eps) {
						helpfulArray[startDataIndex + k] = cluster;
						helpfulArray[startDataIndex + amountOfPeaks + k] = k;
						++NumNeibor;
					}
				}
				
			}
		}

	return cluster - 1;
}


__device__ void calculateDiscreteModel(double* X, const double* a, const double h)
{
	double h1 = a[0] * h;
	double h2 = (1 - a[0]) * h;
	double cos_term = cosf(a[5] * X[1]);
	X[0] = __fma_rn(h1, (-a[6] * X[1]), X[0]);          // x0 += d_h1 * (-a6 * x1)
	X[1] = __fma_rn(h1, (a[6] * X[0] + a[1] * X[2]), X[1]); // x1 += d_h1 * (a6 * x0 + a1 * x2)
	X[2] = __fma_rn(h1, (a[2] - a[3] * X[2] + a[4] * cos_term), X[2]); // x2 += d_h1 * (a2 - a3 * x2 + a4 * cos_term)

	// Вычисление общего коэффициента для второй фазы
	float inv_den = __frcp_rn(__fmaf_rn(a[3], h2, 1.0f));     // Здесь fused не нужен, так как нет умножения с последующим сложением

	// Вторая фаза
	X[2] = __fma_rn(h2, (a[2] + a[4] * cos_term), X[2]) * inv_den; // x2 = fma(d_h2, (a2 + a4 * cos_term), x2) * inv_den
	X[1] = __fma_rn(h2, (a[6] * X[0] + a[1] * X[2]), X[1]); // x1 += d_h2 * (a6 * x0 + a1 * x2)
	X[0] = __fma_rn(h2, (-a[6] * X[1]), X[0]);          // x0 += d_h2 * (-a6 * x1)
}

__device__ double distance(double x1, double y1, double x2, double y2)
{
	if (x1 == x2 && y1 == y2)
		return 0;
	double dx = x2 - x1;
	double dy = y2 - y1;

	return hypotf(dx, dy);
}

} // Bifurcation_constants
